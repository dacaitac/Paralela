#include <stdio.h>
#include <hip/hip_runtime.h>

#define ITER  10000000000  // Number of bins
#define NUMBLOCKS   13  // Number of thread blocks
#define NUMTHREADS 192  // Number of threads per block
int tid;
float pi;

// Kernel
__global__ void pic(float *sum, int nbin, float step, int nthreads, int nblocks) {
	int i;
	float x;
	int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
	for (i=idx; i< nbin; i+=nthreads*nblocks) {
		x = (i+0.5)*step;
		sum[idx] += 4.0/(1.0+x*x);
	}
}

// Main
int main() {
	dim3 dimGrid(NUMBLOCKS,1,1);  // Grid dimensions
	dim3 dimBlock(NUMTHREADS,1,1);  // Block dimensions

	float *sumHost, *sumDev;  // Pointer to host & device arrays

	float step = 1.0/ITER;  // Step size

	size_t size = NUMBLOCKS*NUMTHREADS*sizeof(float);  //Array memory size

	sumHost = (float *)malloc(size);  //  Allocate array on host

	hipMalloc((void **) &sumDev, size);  // Allocate array on device
	// Initialize array in device to 0
	hipMemset(sumDev, 0, size);
	// Do calculation on device
	pic <<<dimGrid, dimBlock>>> (sumDev, ITER, step, NUMTHREADS, NUMBLOCKS); // call CUDA kernel
	// Retrieve result from device and store it in host array
	hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);
	for(tid=0; tid<NUMTHREADS*NUMBLOCKS; tid++)
		pi += sumHost[tid];
	pi *= step;

	// Print results
	printf("PI = %f\n",pi);

	// Cleanup
	free(sumHost);
	hipFree(sumDev);

	return 0;
}
